#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define N 10

__global__ void add(int *a, int *b, int *c)
{
	int tid = blockIdx.x;
	c[tid] = a[tid] + b[tid];
}

int main()
{
	int a[N], b[N], c[N];
	int *deva, *devb, *devc;
	//��device�Ϸ����ڴ�
	hipMalloc((void **)&deva, N * sizeof(int));
	hipMalloc((void **)&devb, N * sizeof(int));
	hipMalloc((void **)&devc, N * sizeof(int));

	//��host��Ϊ���鸳ֵ
	for (int i = 0; i < N; ++i)
	{
		a[i] = -i;
		b[i] = i * i;
	}

	//�������ֵ������device��
	hipMemcpy(deva, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devb, b, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devc, c, N * sizeof(int), hipMemcpyHostToDevice);

	//����Kernel����
	add <<<N,1 >> >(deva, devb, devc);

	//�������device����host
	hipMemcpy(c, devc, N * sizeof(int), hipMemcpyDeviceToHost);

	//������
	for (size_t i = 0; i < N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	//�ͷ�device������ڴ�
	hipFree(deva);
	hipFree(devb);
	hipFree(devc);

	return 0;
}